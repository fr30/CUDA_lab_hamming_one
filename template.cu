#include "hip/hip_runtime.h"
﻿/*
Jakub Frąc 298 795

Hamming One solved in O(n*k*logn)

It could be solved in O(nk) if instead of using binsearch I used 3 small hashing constants for modulo
and then calculate 3 different hashes for single word. Chance of a collision of such a technique
would be p_1 * p_2 * p_3 where p_i is our i-th prime hash constant.

Both binsearch and calculating prefix sums (or prefix hashes) could be implemented more efficiently
with use of GPU-scan algorithm.
*/

#include <math.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <algorithm>
#include <string>
#include <fstream>
#include <chrono>
#include <numeric>

#define N 100*1001
#define M 1050

using namespace std;

// Hashing prime number constant
__constant__ int d_p = 2137;


// Function to calculate array with hash prefixes for given word
__device__ void calculatePref(unsigned long long *pref, unsigned long long *pPow, const char* word, int strlen) {
	pref[0] = word[0];

	for (int i = 1; word[i] != '\0'; i++)
		pref[i] = pref[i - 1] + word[i] * pPow[i];
}

// Performes binsearch on array of hashes created during preprocessing. 
// If hash given as parameter is found in the array - it returns its index.
__device__ int fIndex(unsigned long long hash, unsigned long long *idx, int n) {
	int l = 0, r = n - 1;
	while (l != r) {
		int mid = (l + r) / 2;
		if (idx[mid] < hash)
			l = mid + 1;
		else
			r = mid;
	}
	if (idx[r] != hash)
		return -1;
	return r;
}

// Function to find index in the hashmap.
__device__ int find(unsigned long long hash, int *hMap, unsigned long long *idx, int n) {
	int id = fIndex(hash, idx, n);
	if (id == -1)
		return -1;

	return hMap[id];
}

// Function to add hash to the hashmap.
__device__ void add(unsigned long long hash, int index, int *hMap, unsigned long long *idx, int n) {
	int id = fIndex(hash, idx, n);
	atomicExch(hMap + id, index);
}

// Each thread receives it's word. First it calculates prefix hashes for given word. It then adds the word to the hashmap.
// Then it starts iterating throughout each letter of a word, calculating hash of word with i-th bit flipped (with hamming dist 1
// It then checks if given hash exists in the hashmap - if it does it adds it as a found pair.
__global__ void Kernel(int *n, char* words, unsigned long long *pPow, int *hMap, int *wlens, int *results, unsigned long long *idx) {
	int tid = blockDim.x * blockIdx.x + threadIdx.x;

	if (tid >= *n)
		return;

	unsigned long long pref[M];
	unsigned long long curHash = 0;
	int wlen = wlens[tid];

	const char *word = (words + tid * M);

	calculatePref(pref, pPow, word, wlen);

	curHash = 0;

	add(pref[wlen - 1], tid, hMap, idx, *n);

	for (int i = wlen - 1; i >= 0; i--) {
		unsigned long long prefHash = 0;
		if (i > 0)
			prefHash = pref[i - 1];

		unsigned long long hammHash = prefHash + ((word[i] - '0' + 1) % 2 + '0') * pPow[i] + curHash * pPow[i + 1];


		int index = find(hammHash, hMap, idx, *n);

		if (index != -1) {
			results[tid] = index;
		}

		curHash = curHash * d_p + word[i];
	}
}

const int h_p = 2137;

unsigned long long h_pPow[M], h_idx[N], h_wHash[N];
int h_hMap[N], h_results[N], h_wlens[N];
char h_words[N][M], h_temp[N][M];

// Calculating hash of a word using honer's algorithm
unsigned long long calcHash(char *word) {
	unsigned long long hash = 0;

	for (int i = strlen(word) - 1; i >= 0; i--)
		hash = hash * h_p + word[i];

	return hash;
}

// Pretty self explanatory
void readInput(int *h_n) {
	ifstream myfile("test.big");

	if (myfile.is_open())
	{
		string line;
		getline(myfile, line);
		*h_n = stoi(line);
		int n = *h_n;

		for (int i = 0; i < n; i++)
			myfile.getline(h_words[i], M);

		myfile.close();
	}
	else {
		cout << "Unable to open file";
		exit(0);
	}
}
// Calculates hash of every word, then sorts the words by their hash (hash with smaller value first)
// We can then do binsearch on these hashes and grab index the word associated with it.
// We also calculate word lengths and array containing next powers of prime number p (declared above)
void preprocessInput(int h_n) {
	for (int i = 0; i < h_n; i++)
		h_wHash[i] = calcHash(h_words[i]);

	iota(h_idx, h_idx + h_n, 0);
	sort(h_idx, h_idx + h_n, [](unsigned long long l, unsigned long long r) { return h_wHash[l] < h_wHash[r]; });

	for (int i = 0; i < h_n; i++)
		memmove(h_temp[i], h_words[h_idx[i]], M);

	for (int i = 0; i < h_n; i++) {
		memmove(h_words[i], h_temp[i], M);
		h_idx[i] = h_wHash[h_idx[i]];
	}

	for (int i = 0; i < h_n; i++)
		h_wlens[i] = strlen(h_words[i]);

	h_pPow[0] = 1;
	for (int i = 1; i < M; i++)
		h_pPow[i] = h_pPow[i - 1] * h_p;

	for (int i = 0; i < N; i++)
		h_hMap[i] = -1;
}
// Pretty self explanatory
void prepareMemory(int h_n, int *&d_results, int *&d_wlens, int *&d_hMap, int *&d_n, unsigned long long *&d_pPow, unsigned long long *&d_idx, char *&d_words) {
	memset(h_results, -1, h_n * sizeof(int));

	hipMalloc(&d_results, h_n * sizeof(int));
	hipMalloc(&d_wlens, h_n * sizeof(int));
	hipMalloc(&d_words, h_n * M * sizeof(char *));
	hipMalloc(&d_n, sizeof(int));
	hipMalloc(&d_pPow, M * sizeof(long long));
	hipMalloc(&d_hMap, h_n * sizeof(int));
	hipMalloc(&d_idx, h_n * sizeof(long long));

	hipMemcpy(d_pPow, h_pPow, h_n * sizeof(long long), hipMemcpyHostToDevice);
	hipMemcpy(d_hMap, h_hMap, h_n * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_idx, h_idx, h_n * sizeof(long long), hipMemcpyHostToDevice);
	hipMemcpy(d_n, &h_n, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_results, h_results, h_n * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_words, h_words, h_n * M * sizeof(char), hipMemcpyHostToDevice);
	hipMemcpy(d_wlens, h_wlens, h_n * sizeof(int), hipMemcpyHostToDevice);
}

// Calculates number of threads, starts kernel and copies result to cpu memory
void runAlgorithm(int h_n, int *d_n, int *d_hMap, int *d_wlens, int *d_results, unsigned long long *d_pPow, unsigned long long *d_idx, char *d_words) {
	int threadsPerBlock = 256;
	int blocksPerGrid = (h_n + threadsPerBlock - 1) / threadsPerBlock;

	Kernel << <blocksPerGrid, threadsPerBlock >> > (d_n, d_words, d_pPow, d_hMap, d_wlens, d_results, d_idx);

	int r = hipDeviceSynchronize();
	if (r != hipSuccess) {
		cout << "error" << " " << r << endl;
		exit(0);
	}
	hipMemcpy(h_results, d_results, h_n * sizeof(int), hipMemcpyDeviceToHost);
}


// Pretty self explanatory
void freeMemory(unsigned long long *d_pPow, unsigned long long *d_idx, int *d_hMap, int *d_wlens, int *d_n, int *d_results, char *d_words) {
	hipFree(d_pPow);
	hipFree(d_idx);
	hipFree(d_hMap);
	hipFree(d_wlens);
	hipFree(d_n);
	hipFree(d_results);
	hipFree(d_words);
}

int main() {
	unsigned long long *d_pPow, *d_idx;
	int h_n, *d_hMap, *d_n, *d_wlens, *d_results;
	char *d_words;

	readInput(&h_n);

	auto gStart = std::chrono::high_resolution_clock::now();
	auto start = std::chrono::high_resolution_clock::now();
	preprocessInput(h_n);
	auto end = std::chrono::high_resolution_clock::now();

	cout << "Time spent for preprocessing input: " << std::chrono::duration_cast<std::chrono::microseconds>(end - start).count() / 1e6 << endl;

	start = std::chrono::high_resolution_clock::now();
	prepareMemory(h_n, d_results, d_wlens, d_hMap, d_n, d_pPow, d_idx, d_words);
	end = std::chrono::high_resolution_clock::now();
	cout << "Time spent for preparation of memory: " << std::chrono::duration_cast<std::chrono::microseconds>(end - start).count() / 1e6 << endl;

	start = std::chrono::high_resolution_clock::now();
	runAlgorithm(h_n, d_n, d_hMap, d_wlens, d_results, d_pPow, d_idx, d_words);
	end = std::chrono::high_resolution_clock::now();
	cout << "Time spent for running the algorithm: " << std::chrono::duration_cast<std::chrono::microseconds>(end - start).count() / 1e6 << endl;

	int ans = 0;
	for (int i = 0; i < h_n; i++) {
		if (h_results[i] > i) {
			ans++;
		}
	}

	auto gEnd = std::chrono::high_resolution_clock::now();
	cout << "Overall time spent: " << std::chrono::duration_cast<std::chrono::microseconds>(gEnd - gStart).count() / 1e6 << endl;

	cout << "Answer: " << ans << endl;
	freeMemory(d_pPow, d_idx, d_hMap, d_wlens, d_n, d_results, d_words);
}

